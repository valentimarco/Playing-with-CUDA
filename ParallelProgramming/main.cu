
#include <hip/hip_runtime.h>
#include <iostream>

#define N 66000

__global__ void add(int* a, int* b, int* c){
    int tid = blockIdx.x; // block index 
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}


int main(){
    int a[N], b[N], c[N] = {0};
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    for(int i=0; i < N; i++){
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(dev_a,a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c,dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i < N; i++){
        std::cout << a[i] << " + " << b[i] << "= "<< c[i] << std::endl;
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}