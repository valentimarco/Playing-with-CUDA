
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void add(int a, int b, int *c){
    *c = a + b;
}   
    


int main(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    std::cout << "Hello World! You have " << deviceCount << " available device" << std::endl;

    std::cout << "Info for each device: " << std::endl;

    hipDeviceProp_t deviceProp;

    for (int i = 0; i < deviceCount; i++) {
        hipGetDeviceProperties(&deviceProp, i);
        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "TotalGlobalMem" << ": " << deviceProp.totalGlobalMem << "Bytes"<< std::endl;
        std::cout << "SharedMemPerBlock"<< ": " << deviceProp.sharedMemPerBlock << "Bytes" << std::endl;
    }
    memset(&deviceProp, 0, sizeof(hipDeviceProp_t));
    deviceProp.major = 2;
    deviceProp.minor = 0;
    
    std::cout << "Selecting all gpu with compute capability >= 2.0" << std::endl;
    hipChooseDevice(&deviceCount, &deviceProp);
    hipSetDevice(deviceCount);
    
    

    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    add<<<1,1>>>(2,7,dev_c);
    // host memory can only be access by host code, and vice versa!
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);




    return 0;
}